#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define SPHERES 20 //spheres number to rendering
#define rnd( x ) (x * rand() / RAND_MAX) //macro for making random number
#define INF 2e10f
#define DIM 2048 //image size 2048*2048

struct Sphere {
    float r, b, g;
    float radius;
    float x, y, z;
    //check if ray and sphere hit each other
    //it is a function that can be run at GPU device
    __device__ float hit(float ox, float oy, float *n) {
        float dx = ox - x;
        float dy = oy - y;
        //if pixel value is in sphere
        if (dx*dx + dy*dy < radius*radius) {
            float dz = sqrtf(radius*radius - dx*dx - dy*dy);
            *n = dz / sqrtf(radius * radius);
            return dz + z;
        }
        return -INF;
    }
};

__global__ void cuda_kernel(Sphere* s, unsigned char* ptr) {
    //calculate current thread's 2D location
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= DIM || y >= DIM) return;

    int offset = x + y * DIM;
    float ox = (x - DIM/2);
    float oy = (y - DIM/2);

    float r = 0, g = 0, b = 0;
    float maxz = -INF;

    for(int i = 0; i < SPHERES; i++) {
        float n;
        float t = s[i].hit(ox, oy, &n);
        //if present sphere is more closer
        if (t > maxz) {
            float fscale = n;
            r = s[i].r * fscale;
            g = s[i].g * fscale;
            b = s[i].b * fscale;
            maxz = t;
        }
    }
    //calculated color value -> 0~225 bitmap
    ptr[offset*4 + 0] = (int)(r * 255);
    ptr[offset*4 + 1] = (int)(g * 255);
    ptr[offset*4 + 2] = (int)(b * 255);
    ptr[offset*4 + 3] = 255;
}
//bitmap data -> ppm image file
void ppm_write(unsigned char* bitmap, int xdim, int ydim, const char* filename) {
    FILE* fp = fopen(filename, "w");
    if (!fp) {
        printf("Error: Cannot create file %s\n", filename);
        return;
    }

    fprintf(fp, "P3\n");
    fprintf(fp, "%d %d\n", xdim, ydim);
    fprintf(fp, "255\n");

    for (int y = 0; y < ydim; y++) {
        for (int x = 0; x < xdim; x++) {
            int i = x + y * xdim;
            fprintf(fp, "%d %d %d ", bitmap[4*i], bitmap[4*i+1], bitmap[4*i+2]);
        }
        fprintf(fp, "\n");
    }
    fclose(fp);
}
//Check if error occured after call CUDA API
#define CUDA_CHECK(call) \
    do { \
        hipError_t error = call; \
        if (error != hipSuccess) { \
            printf("CUDA error at %s:%d - %s\n", __FILE__, __LINE__, hipGetErrorString(error)); \
            exit(1); \
        } \
    } while(0)

int main() {
    srand(time(NULL));

    Sphere *h_spheres = (Sphere*)malloc(sizeof(Sphere) * SPHERES);
    unsigned char *h_bitmap = (unsigned char*)malloc(sizeof(unsigned char) * DIM * DIM * 4);

    for (int i = 0; i < SPHERES; i++) {
        h_spheres[i].r = rnd(1.0f);
        h_spheres[i].g = rnd(1.0f);
        h_spheres[i].b = rnd(1.0f);
        h_spheres[i].x = rnd(2000.0f) - 1000;
        h_spheres[i].y = rnd(2000.0f) - 1000;
        h_spheres[i].z = rnd(2000.0f) - 1000;
        h_spheres[i].radius = rnd(200.0f) + 40;
    }
    //GPU memory pointer
    Sphere *d_spheres;
    unsigned char *d_bitmap;
    //allocate GPU memory
    CUDA_CHECK(hipMalloc((void**)&d_spheres, sizeof(Sphere) * SPHERES));
    CUDA_CHECK(hipMalloc((void**)&d_bitmap, sizeof(unsigned char) * DIM * DIM * 4));
    //copy SPHERE data host -> device
    CUDA_CHECK(hipMemcpy(d_spheres, h_spheres, sizeof(Sphere) * SPHERES, hipMemcpyHostToDevice));

    dim3 blockSize(16, 16);  // 16x16 threads per block
    dim3 gridSize((DIM + blockSize.x - 1) / blockSize.x, (DIM + blockSize.y - 1) / blockSize.y);
    //calculate execution time
    clock_t start_time = clock();
    //run CUDA kernel 
    cuda_kernel<<<gridSize, blockSize>>>(d_spheres, d_bitmap);
    CUDA_CHECK(hipGetLastError());
    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(h_bitmap, d_bitmap, sizeof(unsigned char) * DIM * DIM * 4, hipMemcpyDeviceToHost));
    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;

    printf("CUDA ray tracing: %.3f sec\n", elapsed_time);

    ppm_write(h_bitmap, DIM, DIM, "result2.ppm");
    printf("[result2.ppm] was generated.\n");
    CUDA_CHECK(hipFree(d_spheres));
    CUDA_CHECK(hipFree(d_bitmap));
    free(h_spheres);
    free(h_bitmap);

    return 0;
}
