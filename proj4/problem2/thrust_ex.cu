#include "hip/hip_runtime.h"
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/iterator/transform_iterator.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>

// Function object for computing 4/(1+x^2) at each step
struct pi_functor {
    const double step;

    pi_functor(double _step) : step(_step) {}

    __host__ __device__
    double operator()(const long& i) const {
        double x = (i + 0.5) * step;
        return 4.0 / (1.0 + x * x);
    }
};

int main() {
    const long num_steps = 1000000000L;  // 1 billion steps
    const double step = 1.0 / (double)num_steps;

    printf("Thrust Pi Calculation Started...\n");
    printf("Number of steps: %ld\n", num_steps);

    // Start timing
    clock_t start_time = clock();

    // Create counting iterator from 0 to num_steps-1
    thrust::counting_iterator<long> first(0);
    thrust::counting_iterator<long> last = first + num_steps;

    // Create transform iterator that applies pi_functor to each index
    thrust::transform_iterator<pi_functor, thrust::counting_iterator<long>>
        transform_first(first, pi_functor(step));
    thrust::transform_iterator<pi_functor, thrust::counting_iterator<long>>
        transform_last(last, pi_functor(step));

    // Sum all the transformed values using thrust::reduce
    double sum = thrust::reduce(transform_first, transform_last, 0.0, thrust::plus<double>());

    // Calculate pi
    double pi = step * sum;

    // End timing
    clock_t end_time = clock();
    double elapsed_time = ((double)(end_time - start_time)) / CLOCKS_PER_SEC;

    // Print results
    printf("Execution Time : %.10lfsec\n", elapsed_time);
    printf("pi=%.10lf\n", pi);

    return 0;
}
